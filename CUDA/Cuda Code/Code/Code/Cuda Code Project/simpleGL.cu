#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
* This code is a hevily modified form of the sampe code from the git hub repository https://github.com/NVIDIA/cuda-samples/tree/master
* This orignal sample can be found in Samples/5_Domain_Specific/simpleGL
* Almost all linker settings and include calls comes from and where set up with that
* And all functions orignally from the sample code has been either removed or heavily modified, and functions that have been slightly
* have been noted with a comment above the function to say its from simpleGL
* If a function has no comment abouve it, then it has either been completly recoded, or has been added out of source
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <helper_gl.h>
#include <GL/freeglut.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>   

// CUDA helper functions
#include <hip/hip_runtime_api.h>        

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 10.0f
#define THRESHOLD 0.30f
#define REFRESH_DELAY 10

const unsigned int windowWidth  = 810;
const unsigned int windowHeight = 500;

const unsigned int N = 20;
const unsigned int M = 20;

GLuint vbo;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

//My values
const float squareSize = 0.01f;
const float gapSize = 0.08571428f;

const float initialMass = 0.0004f;
const float initialDampingCoeff = 0.003f;
const float springCoeff = 4.0f;
const float springRelaxDistance = 0.0956f;
const float distanceThreshold = 0.1f;

int runCount = 0;
std::vector<int> secondsToDisplay;

using clock2 = std::chrono::high_resolution_clock;

// Declare global variables
std::chrono::time_point<clock2> startTime;
std::chrono::time_point<clock2> endTime;
std::chrono::duration<double> deltaTime;

std::chrono::time_point<clock2> testTimer;

bool gravityEnabled = false;
bool visualsEnabled = false;
bool windEnabled = false;
bool testEnabled = false;

struct Point {
    float2 position;
    float2 prevPosition;
    float2 velocity;
    float2 externalForce;
    float mass;
    float dampingCoeff;
    int adjPoints[4];
    bool hasPhysics;
};

std::vector<Point> points;
std::vector<float> vertices;
std::vector<std::pair<int, int>> connections;

//End of my values

// mouse controls
int mouseOldX, mouseOldY;
int mouseButtons = 0;
float rotateX = 0.0, rotateY = 0.0;
float translateZ = -3.0;

StopWatchInterface *timer = NULL;

int fpsCount = 0;        
int fpsLimit = 1;        
int gIndex = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int gTotalErrors = 0;
bool gbQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a,b) ((a > b) ? a : b)

bool runSoftbody(int argc, char **argv, char *ref_file);
void cleanup();

bool initGL(int *argc, char **argv);
void createVBO(GLuint *vbo);
void deleteVBO(GLuint *vbo);

void display();
void keyboard(unsigned char key, int x, int y);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void timerEvent(int value);

void runCuda();

void updateVBO();
void drawConnections();
void setupVertices();
void setupPoints();

const char *sSDKsample = "Cuda Softbody Simulation";

__global__ void updatePositions(Point* points, int width, int height, bool gravityEnabled, float springCoeff, float springRelaxDistance,float deltaTime)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * width + idx;

    if (idx < width && idy < height)
    {
        Point& p = points[index];

        if (p.hasPhysics == true)
        {
            float forceX = 0.0f;
            float forceY = 0.0f;


            for (int j = 0; j < 4; ++j)
            {
                int adj_index = p.adjPoints[j];
                if (adj_index != -1)
                {
                    Point& adj_point = points[adj_index];

                    float dx = adj_point.position.x - p.position.x;
                    float dy = adj_point.position.y - p.position.y;
                    float distance = sqrtf((dx * dx) + (dy * dy));
                    float magnitude = springCoeff * (distance - springRelaxDistance);
                    forceX += magnitude * dx / distance;
                    forceY += magnitude * dy / distance;

                }
            }


            forceX += -p.velocity.x * p.dampingCoeff;
            forceY += -p.velocity.y * p.dampingCoeff;

            if (gravityEnabled)
            {
                forceY += -9.81f * p.mass;
            }

            hiprandState_t state;
            hiprand_init(1234, idx, 0, &state);

            float randomNumber = 0.0f;

            if (p.externalForce.x != 0.0)
            {

                randomNumber = hiprand_uniform(&state);
                //randomNumber = randomNumber * 2.0f - 1.0f;

                forceX += randomNumber * p.externalForce.x;
            }

            if (p.externalForce.y != 0.0)
            {
                randomNumber = hiprand_uniform(&state);
                //randomNumber = randomNumber * 2.0f - 1.0f;

                forceY += randomNumber * p.externalForce.y;
            }

            float accelX = forceX / p.mass;
            float accelY = forceY / p.mass;

            float newPosX = p.position.x + p.velocity.x * deltaTime + 0.5f * accelX * (deltaTime * deltaTime);
            float newPosY = p.position.y + p.velocity.y * deltaTime + 0.5f * accelY * (deltaTime * deltaTime);

            __syncthreads();

            p.prevPosition = p.position;

            p.velocity.x = (newPosX - p.prevPosition.x) / deltaTime;
            p.velocity.y = (newPosY - p.prevPosition.y) / deltaTime;

            p.position.x = newPosX;
            p.position.y = newPosY;

            p.externalForce.x = 0;
            p.externalForce.y = 0;
        }
    }
}

// Main has barely been chaanged from simpleGL
int main(int argc, char **argv)
{
    startTime = clock2::now();
    testTimer = clock2::now();

    for (int i = 0; i <= 300; i += 10) {
        secondsToDisplay.push_back(i);
    }

    char *ref_file = NULL;

    pArgc = &argc;
    pArgv = argv;

    printf("%s starting...\n", sSDKsample);

    if (argc > 1)
    {
        if (checkCmdLineFlag(argc, (const char **)argv, "file"))
        {
            getCmdLineArgumentString(argc, (const char **)argv, "file", (char **)&ref_file);
        }
    }

    printf("\n");

    runSoftbody(argc, argv, ref_file);

    printf("%s completed, returned %s\n", sSDKsample, (gTotalErrors == 0) ? "OK" : "ERROR!");
    exit(gTotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

// computeFPS has barely been chaanged from simpleGL
void computeFPS()
{
    frameCount++;
    fpsCount++;

    if (fpsCount == fpsLimit)
    {
        avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
        fpsCount = 0;
        fpsLimit = (int)MAX(avgFPS, 1.f);

        sdkResetTimer(&timer);
    }

    char fps[256];
    sprintf(fps, "Cuda Softbody Simulation: %3.1f fps", avgFPS);
    glutSetWindowTitle(fps);
}

// computeFPS has barely been chaanged from simpleGL however some extra function calls have been added
bool initGL(int *argc, char **argv)
{
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(windowWidth, windowHeight);
    glutCreateWindow("Cuda Softbody Simulation");
    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMotionFunc(motion);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);

    if (! isGLVersionSupported(2,0))
    {
        fprintf(stderr, "ERROR: Support for necessary OpenGL extensions missing.");
        fflush(stderr);
        return false;
    }

    glClearColor(0.0, 0.0, 0.0, 1.0);
    glDisable(GL_DEPTH_TEST);

    glViewport(0, 0, windowWidth, windowHeight);

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluPerspective(60.0, (GLfloat)windowWidth / (GLfloat) windowHeight, 0.1, 10.0);

    SDK_CHECK_ERROR_GL();

    return true;
}

// This has been modified to remove a set of unused code form orignal sample and has been renamed, oringal name was runCudaTests
bool runSoftbody(int argc, char **argv, char *ref_file)
{
    sdkCreateTimer(&timer);

    int devID = findCudaDevice(argc, (const char **)argv);

    if (ref_file != NULL)
    {

    }
    else
    {
        if (false == initGL(&argc, argv))
        {
            return false;
        }

        glutDisplayFunc(display);
        glutKeyboardFunc(keyboard);
        glutMouseFunc(mouse);
        glutMotionFunc(motion);
        glutCloseFunc(cleanup);


        setupPoints();

        createVBO(&vbo);

        runCuda();

        glutMainLoop();
    }

    return true;
}

void runCuda()
{
	for (int i = 1; i <= 10; ++i) {

		size_t num_bytes;
		Point* dPoints;
		checkCudaErrors(hipMalloc(&dPoints, points.size() * sizeof(Point)));
		checkCudaErrors(hipMemcpy(dPoints, points.data(), points.size() * sizeof(Point), hipMemcpyHostToDevice));

		dim3 block(12, 12);
		dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);


		endTime = clock2::now();

		std::chrono::duration<double> deltaTime = endTime - startTime;
		float timePassed = deltaTime.count();

		if (timePassed > 0.06f)
		{
			timePassed = 0.01f;
		}
		updatePositions << <grid, block >> > (dPoints, N, M, gravityEnabled, springCoeff, springRelaxDistance, timePassed);

		startTime = clock2::now();

		checkCudaErrors(hipMemcpy(points.data(), dPoints, points.size() * sizeof(Point), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(dPoints));

        if (testEnabled)
        {
            std::chrono::duration<double> time = clock2::now() - testTimer;
            runCount++;

            std::cout << "ACPS :" << runCount / time.count() << std::endl;
        }
	}
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle,filename,mode) (fHandle = fopen(filename, mode))
#endif
#endif

//This function has been untouched from source code
void sdkDumpBin2(void *data, unsigned int bytes, const char *filename)
{
    printf("sdkDumpBin: <%s>\n", filename);
    FILE *fp;
    FOPEN(fp, filename, "wb");
    fwrite(data, bytes, 1, fp);
    fflush(fp);
    fclose(fp);
}

void createVBO(GLuint* vbo)
{
    assert(vbo);

    glGenBuffers(1, vbo);
    glBindBuffer(GL_ARRAY_BUFFER, *vbo);

    setupVertices();

    unsigned int numVertices = N * M * 6;
    unsigned int size = numVertices * 4 * sizeof(float);

    glBufferData(GL_ARRAY_BUFFER, size, vertices.data(), GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    SDK_CHECK_ERROR_GL();
}



void updateVBO()
{
    int vertIndex = 0;
    for (unsigned int i = 0; i < M; ++i)
    {
        for (unsigned int j = 0; j < N; ++j)
        {
            float x0 = points[i * N + j].position.x;
            float y0 = points[i * N + j].position.y;
            float x1 = x0 + squareSize;
            float y1 = y0 + squareSize;

            vertices[vertIndex++] = x0; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x0; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;

            vertices[vertIndex++] = x0; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
        }
    }

    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, vertices.size() * sizeof(float), vertices.data(), GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void setupVertices()
{
    unsigned int numVertices = N * M * 6;
    vertices.resize(numVertices * 4);

    float totalGapWidth = 2.0f - (N * squareSize);
    float totalGapHeight = 2.0f - (M * squareSize);
    float gapX = totalGapWidth / (N + 1);
    float gapY = totalGapHeight / (M + 1);

    int vertIndex = 0;

    for (unsigned int i = 0; i < M; ++i)
    {
        for (unsigned int j = 0; j < N; ++j)
        {
            float x0 = -1.0f + gapX * (j + 1) + squareSize * j;
            float y0 = -0.5f + gapY * (i + 1) + squareSize * i;

            float x1 = x0 + squareSize;
            float y1 = y0 + squareSize;

            vertices[vertIndex++] = x0; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x0; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;

            vertices[vertIndex++] = x0; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y0; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
            vertices[vertIndex++] = x1; vertices[vertIndex++] = y1; vertices[vertIndex++] = 0.0f; vertices[vertIndex++] = 1.0f;
        }
    }
}

void drawConnections()
{
    glColor3f(1.0f, 0.0f, 0.0f);

    glBegin(GL_QUADS);
    for (const auto& connection : connections)
    {
        const Point& p1 = points[connection.first];
        const Point& p2 = points[connection.second];

        float dx = p2.position.x - p1.position.x;
        float dy = p2.position.y - p1.position.y;
        float distance = sqrt(dx * dx + dy * dy);

        float unitX = dx / distance;
        float unitY = dy / distance;

        float perpX = -unitY;
        float perpY = unitX;

        float offsetX = perpX * squareSize / 2;
        float offsetY = perpY * squareSize / 2;

        float x1 = p1.position.x - offsetX;
        float y1 = p1.position.y - offsetY;
        float x2 = p1.position.x + offsetX;
        float y2 = p1.position.y + offsetY;
        float x3 = p2.position.x + offsetX;
        float y3 = p2.position.y + offsetY;
        float x4 = p2.position.x - offsetX;
        float y4 = p2.position.y - offsetY;

        glVertex3f(x1, y1, 0.0f);
        glVertex3f(x2, y2, 0.0f);
        glVertex3f(x3, y3, 0.0f);
        glVertex3f(x4, y4, 0.0f);
    }
    glEnd();
}

void setupPoints()
{
    points.resize(N * M);

    //float totalGapWidth = 2.0f - (N * squareSize);
    //float totalGapHeight = 2.0f - (M * squareSize);
    //float gapX = totalGapWidth / (N + 1);
    //float gapY = totalGapHeight / (M + 1);
    float gapY = gapSize;
    float gapX = gapSize;

    for (unsigned int i = 0; i < M; ++i)
    {
        for (unsigned int j = 0; j < N; ++j)
        {
            float x0 = -1.0f + gapX * (j + 1) + squareSize * j;
            float y0 = -0.5f + gapY * (i + 1) + squareSize * i;

            Point& p = points[i * N + j];
            p.position = make_float2(x0, y0);
            p.prevPosition = p.position;
            p.velocity = make_float2(0.0f, 0.0f);
            p.externalForce = make_float2(0.0f, 0.0f);
            p.mass = initialMass;
            p.dampingCoeff = initialDampingCoeff;
            p.hasPhysics = true;

            if ((i == M - 1 && j == 0) || (i == M - 1 && j == N - 1))
                p.hasPhysics = false;
        }
    }

    for (unsigned int i = 0; i < M; ++i)
    {
        for (unsigned int j = 0; j < N; ++j)
        {
            Point& p = points[i * N + j];
            int adjIndex = 0;
            if (i > 0)
            {
                p.adjPoints[adjIndex++] = (i - 1) * N + j;
            }
            if (i < M - 1)
            {
                p.adjPoints[adjIndex++] = (i + 1) * N + j;
            }
            if (j > 0)
            {
                p.adjPoints[adjIndex++] = i * N + (j - 1);
            }
            if (j < N - 1)
            {
                p.adjPoints[adjIndex++] = i * N + (j + 1);
            }
            for (; adjIndex < 4; ++adjIndex)
            {
                p.adjPoints[adjIndex] = -1;
            }

            for (int k = 0; k < 4; ++k)
            {
                if (p.adjPoints[k] != -1)
                {
                    int adjPointIndex = p.adjPoints[k];
                    if (i * N + j < adjPointIndex)
                    {
                        connections.emplace_back(i * N + j, adjPointIndex);
                    }
                }
            }
        }
    }
}


//This function had code removed that was no longer being used
void deleteVBO(GLuint *vbo)
{
    glBindBuffer(1, *vbo);
    glDeleteBuffers(1, vbo);

    *vbo = 0;
}

//Disply has been added too with extra function calls for new uses
void display()
{
    sdkStartTimer(&timer);

    // run CUDA kernel to generate vertex positions
    if (windEnabled)
    {
        for (auto& point : points)
        {
            point.externalForce.x = 0.03f;
        }
    }

    runCuda();
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    if(visualsEnabled)
    {
    updateVBO();

    

    // set view matrix
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    glTranslatef(0.0, 0.0, translateZ);
    glRotatef(rotateX, 1.0, 0.0, 0.0);
    glRotatef(rotateY, 0.0, 1.0, 0.0);

    // render from the vbo
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glVertexPointer(4, GL_FLOAT, 0, 0);

    glEnableClientState(GL_VERTEX_ARRAY);
    glColor3f(1.0, 0.0, 0.0);
    glDrawArrays(GL_TRIANGLES, 0, (N * M) * 6);
    glDisableClientState(GL_VERTEX_ARRAY);

    drawConnections();
    }
    glutSwapBuffers();

    g_fAnim += 0.01f;

    sdkStopTimer(&timer);
    computeFPS();
}

// This function has been untoched since source
void timerEvent(int value)
{
    if (glutGetWindow())
    {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

// This function has been untoched since source
void cleanup()
{
    sdkDeleteTimer(&timer);

    if (vbo)
    {
        deleteVBO(&vbo);
    }
}

// This function has been added appon for more functions
void keyboard(unsigned char key, int /*x*/, int /*y*/)
{
    switch (key)
    {
    case 'g':
    case 'G':
        gravityEnabled = !gravityEnabled;
        break;
    case 'v':
    case 'V':
        visualsEnabled = !visualsEnabled;
        break;
    case 'w':
    case 'W':
        windEnabled = !windEnabled;
        break;
    case 27:
        glutDestroyWindow(glutGetWindow());
        return;
    }
}

float distance(float2 a, float2 b) {
    return sqrtf((a.x - b.x) * (a.y - b.y) + (a.y - b.y) * (a.y - b.y));
}

float2 normalize(float2 v) {
    float length = sqrtf(v.x * v.x + v.y * v.y);
    if (length != 0) {
        v.x /= length;
        v.y /= length;
    }
    return v;
}

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouseButtons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouseButtons = 0;
    }

    mouseOldX = x;
    mouseOldY = y;
}

void motion(int x, int y)
{
    float worldX = (x / (float)windowWidth) * 2.0f - 1.0f;
    float worldY = 1.0f - (y / (float)windowHeight) * 2.0f;

    float2 mousePos = make_float2(worldX, worldY);

    for (auto& point : points)
    {
        float dist = distance(mousePos, point.position);
        if (dist < distanceThreshold)
        {
            float2 mouse_direction = make_float2(mousePos.x - point.position.x, mousePos.y - point.position.y);
            mouse_direction = normalize(mouse_direction);
            point.externalForce = make_float2(mouse_direction.x * 5.0f, mouse_direction.y * 5.0f);
        }
    }
}
